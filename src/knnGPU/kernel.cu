#include "hip/hip_runtime.h"
#define MAX_NUM_SAMPLES {{ MAX_NUM_SAMPLES }}
#define MAX_NUM_TEST {{ MAX_NUM_TEST }}
#define MAX_NUM_FEATURES {{ MAX_NUM_FEATURES }}
#define K   {{ K }}

/**
 * Compute the euclidean distance between myFeatures and otherFeatures.
 * @param  myFeatures    Float pointer to the characteristics values of the first sample.
 * @param  otherFeatures Float pointer to the characteristics values of the second sample.
 * @param  numFeatures   Number of features that both samples have.
 * @return               The euclidean distance between myFeatures and otherFeatures.
 */
__device__ float computeDistance(float* myFeatures, float* otherFeatures,
								 int numFeatures){
	float distance = 0;

	// Compute the squared euclidean distance.
    for (size_t i = 0; i < numFeatures; i++) {
        distance += (myFeatures[i] - otherFeatures[i]) *
					(myFeatures[i] - otherFeatures[i]);
    }

	// Returns the euclidean distance.
	return sqrt(distance);
}

/**
 * You know, yet another ordering method.
 * @param a Float2 pointer to a pair of floats (where a.x = index of the sample,
 *          a.y = distance to the current considered sample).
 * @param n Number of samples in the array pointed by a.
 */
__device__ void bubble_sort (float2* a, int n) {
    int i, s = 1;
	float2 t;

	// Stuff copied from a village of La Mancha, the name of which I have no
	// desire to call to mind.
    while (s) {
        s = 0;
        for (i = 1; i < n; i++) {
            if (a[i].y < a[i - 1].y) {
                t = a[i];
                a[i] = a[i - 1];
                a[i - 1] = t;
                s = 1;
            }
        }
    }
}

/**
 * Order kNearest array depending on the stored distance.
 * @param kNearest  Float2 pointer to K+1 samples
 * @param newSample [description]
 */
__device__ void updateKNearest(float2* kNearest, float2 newSample){
	// The last (unconsidered element) is the new one
	kNearest[K] = newSample;

	// Ordering :)
	bubble_sort(kNearest, K+1);
}

/**
 * Returns the most repeated element in a sequence, given that there exist
 * an element with absolute majority. It works with k = 3 but not with k > 3.
 * @param  arr  Array of integer elements
 * @param  size Number of elements in the array
 * @return      The most repeated element in arr
 */
__device__ int votingMethod(int* arr, int size) {
    int current_candidate = arr[0], counter = 0, i;
    for (i = size-1; i >= 0; --i) {
        if (current_candidate == arr[i]) {
            ++counter;
        } else if (counter == 0) {
            current_candidate = arr[i];
            ++counter;
        } else {
            --counter;
        }
    }
    return current_candidate;
}

/**
 * Execute leave-one-out and computes the mean ratio of success
 * @param[in]	devSamples	Pointer to an array of size numSamples*numFeatures,
 * 							where the features of the i-th sample are stored in the slice from
 * 							devSamples[i*numFeatures] to devSamples[i*numFeatures + numFeatures]
 * @param[in]	devTarget   Pointer to an array of size numSamples, where the
 * 							i-th value stores the class of the i-th sample.
 * @param[out]	devResult	Pointer to an array of size numSamples, where the i-th sample will
 * 							store a 1 or a 0 depending on the success of the class prediction.
 * @param[in]	numFeatures Number of features of each sample.
 * @param[in]	numSamples  Number of samples in the data.
 */
__global__ void scoreSolution(void *devSamples, void *devTarget,
							  void *devResult, int numFeatures,
							  int numSamples){
  // Pointers to the features, the target and the result: CUDA global memory :(
  float* globalSamples = (float*)devSamples;
  int* globalTarget = (int*)devTarget;
	int* globalResult = (int*)devResult;

  // The sample represented by this thread is the global identifier of the
  // thread
  int sample = blockIdx.x * blockDim.x + threadIdx.x;

	// Stop execution if the sample id is not in the samples range (necessary for
	// generalizing the number of blocks and threads)
	if(sample >= numSamples){
		return;
	}

	// Index of this thread sample features start in the globalSamples array
  int initOfMyFeatures = sample * numFeatures;
  float myFeatures[MAX_NUM_FEATURES];

  // Population of this thread sample features
  for(int i=0; i<numFeatures; i++){
          myFeatures[i] = globalSamples[initOfMyFeatures + i];
  }

	// Aux sample with invalid index and  infinite distance for initializing the
	// K nearest neighbours array.
	float2 inf;
	inf.x = -1;
	inf.y = 99999999;

	// K (plus one in order to ease the update function) nearest neighbours, stored
	// as float2, where:
	// 		x: sample index
	// 		y: distance to the thread sample
	float2 kNearest[K+1];

	// initialization of the K nearest neighbours array
	for (size_t i = 0; i < K+1; i++) {
		kNearest[i] = inf;
	}

	// Loop aux variable for storing each remaining sample
	float2 newSample;

  // Computation of distances between this thread sample and the remaining ones.
  // TODO: Improve the efficiency of this loop: the matrix of distances is symmetric, use that!
  // TODO: Maybe use shared memory to improve efficiency.
  for(int i=0; i<numSamples; i++){
	// Leave one out main behaviour: do not consider this sample in order not
	// to bias the final score
    if(i == sample){
        continue;
    }

		// New sample index and distance to this thread sample
		newSample.x = i;
		newSample.y = computeDistance(myFeatures,
									  globalSamples + i * numFeatures,
									  numFeatures);

		// Check whether this new sample should be in the K nearest neighbours.
		updateKNearest(kNearest, newSample);
	}

	// Array for storing the classes of the K nearest neighbours.
	int classes[K];

	// Populate the classes array with the classes of the K nearest neighbours.
	for (size_t i = 0; i < K; i++) {
		classes[i] = globalTarget[(int)kNearest[i].x];
	}

	// Voting method. Choose the most repeated class in the classes array.
	// TODO: Generalize to k != 3
	int computedClass = votingMethod(classes, K);

	// Check wether the computed class is equal to the stored class in the actual
	// target array. Set to 1 if success, to 0 if failure.
	globalResult[sample] = computedClass == globalTarget[sample] ? 1 : 0;
}

/**
 * Computes the mean ratio of success of test samples.
 * @param[in]	devSamples	Pointer to an array of size numSamples*numFeatures,
 * 							where the features of the i-th sample are stored in the slice from
 * 							devSamples[i*numFeatures] to devSamples[i*numFeatures + numFeatures]
 * @param[in]	devTest	Pointer to an array of size numTest*numFeatures,
 * 							where the features of the i-th test sample are stored in the slice from
 * 							devTest[i*numFeatures] to devTest[(i+1)*numFeatures]
 * @param[in]	devTargetTrain Pointer to an array of size numSamples, where the
 * 							i-th value stores the class of the i-th sample.
 * @param[in]	devTargetTest Pointer to an array of size numTest, where the
 * 							i-th value stores the class of the i-th sample test.
 * @param[out]	devResult	Pointer to an array of size numTest, where the i-th sample will
 * 							store a 1 or a 0 depending on the success of the class prediction.
 * @param[in]	numFeatures Number of features of each sample.
 * @param[in]	numSamples  Number of samples in the data.
 * @param[in]	numSamples  Number of samples test in the data.
 */
__global__ void scoreOut(void *devSamples, void *devTest,
								void *devTargetTrain, void *devTargetTest,
							  void *devResult, int numFeatures,
							  int numSamples, int numTest){
  // Pointers to the features, the target and the result: CUDA global memory :(
  float* globalSamples = (float*)devSamples;
  float* globalTest = (float*)devTest;
  int* globalTargetTrain = (int*)devTargetTrain;
  int* globalTargetTest = (int*)devTargetTest;
	int* globalResult = (int*)devResult;

  // The sample represented by this thread is the global identifier of the
  // thread
  int test = blockIdx.x * blockDim.x + threadIdx.x;

	// Stop execution if the test id is not in the test range (necessary for
	// generalizing the number of blocks and threads)
	if(test >= numTest){
		return;
	}

	// Index of this thread sample features start in the globalTest array
  int initOfMyFeatures = test * numFeatures;
  float myFeatures[MAX_NUM_FEATURES];

  // Population of this thread sample features
  for(int i=0; i<numFeatures; i++){
          myFeatures[i] = globalTest[initOfMyFeatures + i];
  }

	// Aux sample with invalid index and  infinite distance for initializing the
	// K nearest neighbours array.
	float2 inf;
	inf.x = -1;
	inf.y = 99999999;

	// K (plus one in order to ease the update function) nearest neighbours, stored
	// as float2, where:
	// 		x: sample index
	// 		y: distance to the thread sample
	float2 kNearest[K+1];

	// initialization of the K nearest neighbours array
	for (size_t i = 0; i < K+1; i++) {
		kNearest[i] = inf;
	}

	// Loop aux variable for storing each remaining test sample
	float2 newSample;

  // Computation of distances between this thread sample and the remaining ones.
  // TODO: Improve the efficiency of this loop: the matrix of distances is symmetric, use that!
  // TODO: Maybe use shared memory to improve efficiency.
  for(int i=0; i<numSamples; i++){
		// New sample index and distance to this thread sample
		newSample.x = i;
		newSample.y = computeDistance(myFeatures,
									  globalSamples + i * numFeatures,
									  numFeatures);

		// Check whether this new sample should be in the K nearest neighbours.
		updateKNearest(kNearest, newSample);
	}

	// Array for storing the classes of the K nearest neighbours.
	int classes[K];

	// Populate the classes array with the classes of the K nearest neighbours.
	for (size_t i = 0; i < K; i++) {
		classes[i] = globalTargetTrain[(int)kNearest[i].x];
	}

	// Voting method. Choose the most repeated class in the classes array.
	// TODO: Generalize to k != 3
	int computedClass = votingMethod(classes, K);

	// Check wether the computed class is equal to the stored class in the actual
	// target array. Set to 1 if success, to 0 if failure.
	globalResult[test] = computedClass == globalTargetTest[test] ? 1 : 0;
}
